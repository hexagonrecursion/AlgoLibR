#include "hip/hip_runtime.h"
/*
@File    :   sort.cu
@Time    :   2019/09/09 15:10:32
@Author  :   raoqiyu@gmail.com
@Desc    :   Implementing Parallel Radix Sort with CUDA
*/

#include "AlgoLibR/sort/radix_sort.h"
#include <stdio.h>
#include <hip/hip_runtime.h>


namespace AlgoLibR{
namespace sort{
namespace radix_sort{

__global__
void histogram(const unsigned int * const d_in, unsigned int * const d_out, const int nthBit, const size_t numElems){
    //use shared memory to load the whole block data
    extern __shared__ unsigned int tempElems[];

    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int tid = threadIdx.x;

    if(idx < numElems){
        tempElems[tid] = d_in[idx];
        __syncthreads();
        unsigned int nthBin = (tempElems[tid]>>nthBit)&(numBins-1);
        atomicAdd(&d_out[nthBin],1);
    }
}

// Hillis Steele Scan
__global__
void scan(const unsigned int * const d_in, unsigned int *d_out, unsigned int *d_blockLastElems, const size_t numElems){
    // use shared memory to load the whole block data
    extern __shared__ unsigned int temp[];

    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x*blockDim.x + tid;

    if(idx >= numElems){
        return;
    }

    temp[tid] = d_in[idx];
    __syncthreads();

    for(unsigned int stride = 1; stride < blockDim.x; stride <<= 1){
        unsigned int tmpVal = temp[tid];
        __syncthreads();

        if(tid + stride < blockDim.x)
            temp[tid+stride] += tmpVal;
        __syncthreads();
    }

    // exclusive scan
    d_out[idx] = tid > 0 ? temp[tid-1] : 0;

    if(tid  == (blockDim.x-1))
        d_blockLastElems[blockIdx.x] = temp[tid];
}

__global__
void add(const unsigned int * const d_in, unsigned int * const d_out, const size_t numElems){
    unsigned int bIdx = blockIdx.x;
    unsigned int idx = bIdx*blockDim.x + threadIdx.x;

    // The last element of the previous block should be added to current block's element
    if (idx < numElems){
        d_out[idx]  += d_in[bIdx];
    }

}


// Exclusive Prefix Sum of Histogram(support a large array, not just in one block)
// 1) first do scan on each block
// 2) then do scan one all block's last elements
// 3) then add block's last element to its next block
void prefixSum(unsigned int *d_in, unsigned int *d_out, const size_t numElems){
    const dim3 blockSize(min(1024, (int)numElems));
    const dim3 gridSize(ceil((float)numElems/blockSize.x));


    unsigned int *d_blockLastElems;
    checkCudaErrors(hipMalloc((void**)&d_blockLastElems, gridSize.x*sizeof(unsigned int)));

    // 1) first do scan on each block
    scan<<<gridSize, blockSize, blockSize.x*sizeof(unsigned int)>>>(d_in, d_out, d_blockLastElems, numElems);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    if(gridSize.x > 1){
        // 2) then do scan one all block's last elements
        prefixSum(d_blockLastElems, d_blockLastElems, gridSize.x);
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

        // 3) then add block's last element to its next block
        add<<<gridSize, blockSize>>>(d_blockLastElems, d_out, numElems);
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    }
    checkCudaErrors(hipFree(d_blockLastElems));
}

__global__
void map(const unsigned int * const d_in, unsigned int * const d_out, const size_t numElems, const int mask, const int nthBit){
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx < numElems){
        d_out[idx] = ((d_in[idx]>>nthBit)&(numBins-1)) == mask;
    }
}
__global__
void movebyBins(unsigned int* const d_inputVals,unsigned int* const d_outputVals,
          unsigned int* const d_binElems, unsigned int *d_binScan , unsigned int *d_binHistogram, const size_t numElems, const int mask){
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if(idx < numElems && d_binElems[idx] == 1){
        unsigned int outputIdx = d_binHistogram[mask] + d_binScan[idx];
        d_outputVals[outputIdx] = d_inputVals[idx];
    }
}

void radix_sort_kernel(unsigned int*  h_inputVals,
               unsigned int*  h_outputVals,
               const size_t numElems){

    const dim3 blockSize(1024);
    const dim3 gridSize(ceil((float)numElems/1024));

    unsigned int *d_binHistogram, *d_binScan, *d_binElems, *d_inputVals,*d_outputVals;
    checkCudaErrors(hipMalloc((void **)&d_inputVals, numElems* sizeof(unsigned int)));
    checkCudaErrors(hipMalloc((void **)&d_outputVals, numElems* sizeof(unsigned int)));

    checkCudaErrors(hipMalloc((void**)&d_binHistogram, numBins*sizeof(unsigned int)));
    checkCudaErrors(hipMalloc((void**)&d_binScan, numElems*sizeof(unsigned int)));
    checkCudaErrors(hipMalloc((void**)&d_binElems, numElems*sizeof(unsigned int)));

    checkCudaErrors(hipMemset(d_inputVals,0, numElems*sizeof(unsigned int)));
    checkCudaErrors(hipMemset(d_outputVals,0, numElems*sizeof(unsigned int)));
    checkCudaErrors(hipMemset(d_binHistogram,0, numBins*sizeof(unsigned int)));
    checkCudaErrors(hipMemset(d_binScan,0, numElems*sizeof(unsigned int)));
    checkCudaErrors(hipMemset(d_binElems,0, numElems*sizeof(unsigned int)));

    checkCudaErrors(hipMemcpy(d_inputVals, h_inputVals, numElems * sizeof(unsigned int), hipMemcpyHostToDevice));

    for(int i = 0; i < 8 * (int)sizeof(unsigned int); i += numBits){
        checkCudaErrors(hipMemset(d_binHistogram,0, numBins*sizeof(unsigned int)));

        // 1) Histogram of the number of occurrences of the i-th bit
        histogram<<<gridSize, blockSize, blockSize.x*sizeof(unsigned int)>>>(d_inputVals, d_binHistogram, i, numElems);
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

        // 2) Exclusive Prefix Sum of Histogram
        prefixSum(d_binHistogram, d_binHistogram, numBins);
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

        // 3) Determine relative offset of each digit
        // 4) Combine the results of steps 2 & 3 to determine the final output location for each element and move it there
        for(int j = 0; j < numBins; j++){
            map<<<gridSize, blockSize>>>(d_inputVals, d_binElems, numElems, j, i);
            hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

            prefixSum(d_binElems, d_binScan, numElems);
            movebyBins<<<gridSize,blockSize>>>(d_inputVals, d_outputVals, d_binElems,d_binScan,
                                                d_binHistogram, numElems, j);
            hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
        }
        std::swap(d_inputVals, d_outputVals);

    }
    checkCudaErrors(hipMemcpy(h_outputVals, d_inputVals, numElems*sizeof(unsigned int), hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(d_binHistogram));
    checkCudaErrors(hipFree(d_binScan));
    checkCudaErrors(hipFree(d_binElems));
    checkCudaErrors(hipFree(d_inputVals));
    checkCudaErrors(hipFree(d_outputVals));
}

} // namespace radix_sort
} // namespace sort
} // namespace AlgoLibR