#include "common/utils.h"
#include "sort/sort.h"
#include <stdio.h>
#define N 10


int main(int argc, char *argv[]){
    unsigned int arrs[N] = {5,14,2,16,21,651,61,21,65,16};
    unsigned int h_outs[N];
    unsigned int *d_in, *d_out;

    hipMalloc((void **)&d_in, N* sizeof(unsigned int));
    hipMalloc((void **)&d_out, N* sizeof(unsigned int));
    checkCudaErrors(hipGetLastError());

    hipMemcpy(d_in, arrs, N * sizeof(unsigned int), hipMemcpyHostToDevice);
    checkCudaErrors(hipGetLastError());

    SORT::radix_sort(d_in, d_out,  (size_t) N);

    hipMemcpy(h_outs,d_out, N * sizeof(unsigned int), hipMemcpyDeviceToHost);

    for(int i = 0; i < N; i++){
        printf("%d ",h_outs[i]);
    }
    printf("\n");

    checkCudaErrors(hipFree(d_in));
    checkCudaErrors(hipFree(d_out));
}
