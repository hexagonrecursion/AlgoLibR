#include "common/utils.h"
#include "sort/sort.h"
#include <stdio.h>
#define N 10


int main(int argc, char *argv[]){
    unsigned int arrs[N] = {5,14,2,16,21,651,61,21,65,16};
    unsigned int h_outs[N];
    unsigned int *d_in;

    hipMalloc((void **)&d_in, N* sizeof(unsigned int));
    checkCudaErrors(hipGetLastError());

    hipMemcpy(d_in, arrs, N * sizeof(unsigned int), hipMemcpyHostToDevice);
    checkCudaErrors(hipGetLastError());

    SORT::radix_sort(d_in, h_outs,  (size_t) N);


    for(int i = 0; i < N; i++){
        printf("%d ",h_outs[i]);
    }
    printf("\n");

    checkCudaErrors(hipFree(d_in));
}
